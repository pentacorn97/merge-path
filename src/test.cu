#include <random>
#include <iostream>
#include "../inc/alg.cuh"

int main(int argc, char* argv[])
{
    std::default_random_engine generator;
    std::uniform_int_distribution<int> gen(0, 1000);

    // Test the int type.
    constexpr size_t size_a = 16;
    constexpr size_t size_b = 12;
    constexpr size_t size_ttl = size_a + size_b;
    int arr_a[size_a] = {}, arr_b[size_b] = {}, arr_m[size_ttl] = {};

    for(size_t i=0; i<size_a; ++i) { arr_a[i] = gen(generator); }
    for(size_t i=0; i<size_b; ++i) { arr_b[i] = gen(generator); }
    std::sort(arr_a, arr_a+size_a);
    std::sort(arr_b, arr_b+size_b);

    std::cout << "a: ";
    for(size_t i=0; i<size_a; ++i) { std::cout << arr_a[i] << " "; }
    std::cout << std::endl;
    std::cout << "b: ";
    for(size_t i=0; i<size_b; ++i) { std::cout << arr_b[i] << " "; }
    std::cout << std::endl;

    int *p_a = nullptr, *p_b = nullptr, *p_m = nullptr;
    hipMalloc(&p_a, size_a*sizeof(int));
    hipMalloc(&p_b, size_b*sizeof(int));
    hipMalloc(&p_m, size_ttl*sizeof(int));
    hipMemcpy(p_a, arr_a, size_a*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(p_b, arr_b, size_b*sizeof(int), hipMemcpyHostToDevice);

    std::cout << "Merge (a, b)" << std::endl;
    std::fill(arr_m, arr_m+size_ttl, 0);
    merge::merge_small_k<int><<<1, 32>>>(p_m, p_a, p_b, size_a, size_b);
    hipMemcpy(arr_m, p_m, size_ttl*sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "m: ";
    for(size_t i=0; i<size_ttl; ++i) 
    { std::cout << arr_m[i] << " "; }
    std::cout << (std::is_sorted(arr_m, arr_m+size_ttl) ? "well sorted." : "not sorted.") << std::endl;

    // Test b first.
    std::cout << "Merge (b, a)" << std::endl;
    merge::merge_small_k<int><<<1, 32>>>(p_m, p_b, p_a, size_b, size_a);
    hipMemcpy(arr_m, p_m, size_ttl*sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "m: ";
    for(size_t i=0; i<size_ttl; ++i) { std::cout << arr_m[i] << " "; }
    std::cout << (std::is_sorted(arr_m, arr_m+size_ttl) ? "well sorted." : "not sorted.") << std::endl;

    return 0;
}